﻿
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "math.h"
/*8 bits --> 255 valores*/
#define n 16
#define iterCPU 100

void Calc_Hist(unsigned int* Hist, unsigned int* Image) {
	/*Hacemos el histograma*/
	for (int i = 0; i < n * n; i++) {
		Hist[Image[i]] += 1;
	}

}

int main() {
	unsigned int *Image,*Hist;
	Image = (unsigned int*)malloc(n * n * sizeof(unsigned int));
	Hist = (unsigned int*)malloc( 256* sizeof(unsigned int));
	/*Inicializamos variables*/
	for (int i = 0; i < 256; i++) {
		Hist[i] = 0;//Reemplazo de calloc
	}
	/*Inicializamos la imagen con numeros aleatorios*/
	srand(time(NULL));
	for (int i=0; i < n * n; i++) {
		Image[i] = (unsigned int)(rand()%256);
	}
	/*Medimos tiempo*/
	clock_t startCPU;
	clock_t finishCPU;
	startCPU = clock();
	for (int i = 0; i < iterCPU; i++) {
		Calc_Hist(Hist, Image);
	}
	finishCPU = clock();
	printf("Numero de iteraciones:%d\n", iterCPU);
	printf("CPU serial time: %fms\n", (double)(finishCPU - startCPU) / iterCPU);/// CLK_TCK);

	FILE* archivo;
	archivo = fopen("Histograma.csv", "w+");
	fprintf(archivo, "valores\n");
	for (unsigned int i = 0; i < 256; i++) {
		fprintf(archivo, "%d\n", Hist[i]);
	}
	fclose(archivo);
	printf("Se creo el archivo de histograma");
}